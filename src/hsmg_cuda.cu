#define LX1 10
#define LY1 10
#define LZ1 10

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void h1mg_tnsr3d_kernel(double* __restrict__ v,const int nv, const double* __restrict__ u,const int nu, const double* __restrict__ A, const double* __restrict__ Bt, const double* __restrict__ Ct){
      __shared__ double work1[(LX1+2)*(LX1+2)*(LX1+2)];
      __shared__ double work2[(LX1+2)*(LX1+2)*(LX1+2)];
      int e,i,j,k,l;
      e = blockIdx.x;
      i = threadIdx.x;
      j = threadIdx.y;
      
      if(j < nu){
        for( k = 0; k < nu; ++k){
          work1[i+nv*j+nv*nu*k] = 0.0;
          for(l = 0; l < nu; ++l){
            work1[i+nv*j+nv*nu*k] += A[i+nv*l]*u[l+j*nu+k*nu*nu+e*nu*nu*nu];
          }
        }
      }
      __syncthreads();
      for( k = 0; k< nu; ++k){
        work2[i+nv*j+nv*nv*k] = 0.0;
        for( l = 0; l < nu; ++l){
          work2[i+j*nv + k*nv*nv] += work1[i + l*nv + k*nv*nu]*Bt[l + j*nu];  
        }
      } 
      __syncthreads();
      for( k = 0; k< nv; ++k){
        v[i+j*nv+k*nv*nv+e*nv*nv*nv] = 0.0;
        for( l = 0; l < nu; ++l){
          v[i+j*nv+k*nv*nv+e*nv*nv*nv] += work2[i+j*nv+l*nv*nv]*Ct[l+k*nu];
        }
      }
}

  __global__ void h1mg_do_fast_kernel(double* __restrict__  e, double* __restrict__ r,const double* __restrict__ s, const double* __restrict__ d, const int nl){
      __shared__ double work1[(LX1+2)*(LX1+2)*(LX1+2)];
      __shared__ double work2[(LX1+2)*(LX1+2)*(LX1+2)];
      int el,i,j,k,l,nu,nv,nl2,nel,nn,snel;
      el = blockIdx.x;
      i = threadIdx.x;
      j = threadIdx.y;
      nu = nl;
      nv = nl;
      nn = nl*nl*nl;
      nl2 = nl*nl;
      nel = el*nn;
      snel = 6*el*nl2;

      for( k = 0; k < nu; ++k){
        work1[i+nv*j+nv*nu*k] = 0.0;
        for(l = 0; l < nu; ++l){
          work1[i+nv*j+nv*nu*k] += s[i+nv*l+1*nl2 + 0*nl2*2 + snel]*r[l+j*nu+k*nu*nu+nel];
        }
      }
      
      __syncthreads();
      for( k = 0; k< nu; ++k){
        work2[i+nv*j+nv*nv*k] = 0.0;
        for( l = 0; l < nu; ++l){
          work2[i+j*nv + k*nv*nv] += work1[i + l*nv + k*nv*nu]*s[l + j*nu+0*nl2 + 1*nl2*2 + snel];  
        }
      } 
      __syncthreads();
      for( k = 0; k< nv; ++k){
        e[i+j*nv+k*nv*nv+nel] = 0.0;
        for( l = 0; l < nu; ++l){
          e[i+j*nv+k*nv*nv+nel] += work2[i+j*nv+l*nv*nv]*s[l+k*nu+0*nl2 + 2*nl2*2 + snel];
        }
      }
      for( k = 0; k< nl; ++k){
        r[i+j*nv+k*nl2+nel] = d[i+j*nl+k*nl2+nel]*e[i+j*nv+k*nv*nv+nel]; 
      }
      __syncthreads();
      for( k = 0; k < nu; ++k){
        work1[i+nv*j+nv*nu*k] = 0.0;
        for(l = 0; l < nu; ++l){
          work1[i+nv*j+nv*nu*k] += s[i+nv*l+0*nl2 + 0*nl2*2 + snel]*r[l+j*nu+k*nu*nu+nel];
        }
      }
      __syncthreads();
      for( k = 0; k< nu; ++k){
        work2[i+nv*j+nv*nv*k] = 0.0;
        for( l = 0; l < nu; ++l){
          work2[i+j*nv + k*nv*nv] += work1[i + l*nv + k*nv*nu]*s[l + j*nu+1*nl2 + 1*nl2*2 + snel];  
        }
      } 
      __syncthreads();
      for( k = 0; k< nv; ++k){
        e[i+j*nv+k*nv*nv+nel] = 0.0;
        for( l = 0; l < nu; ++l){
          e[i+j*nv+k*nv*nv+nel] += work2[i+j*nv+l*nv*nv]*s[l+k*nu+1*nl2 + 2*nl2*2 + snel];
        }
      }
}


extern "C" {
  void h1mg_tnsr3d_cuda_(double* __restrict__ v,const int* nv, const double* __restrict__ u,const int* nu, const double* __restrict__ A, const double* __restrict__ Bt, const double* __restrict__ Ct, const int *nel){
       h1mg_tnsr3d_kernel<<<*nel,dim3(*nv,*nv,1)>>>(v,*nv,  u,*nu, A, Bt, Ct);
}  

  void h1mg_do_fast_cuda_(double* __restrict__  e, double* __restrict__ r,const double* __restrict__ s, const double* __restrict__ d, const int* nl, const int *nel){
       h1mg_do_fast_kernel<<<*nel,dim3(*nl,*nl,1)>>>(e,r,s,d,*nl);
}
}
