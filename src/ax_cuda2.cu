#include "hip/hip_runtime.h"
#define LX1 10
#define LY1 10
#define LZ1 10
#include <stdio.h>
#include "nvmlPower.hpp"
__global__ void ax_cuda2_kernel(double* __restrict__ w, const double* __restrict__ u, const double* __restrict__ gxyz, const double* __restrict__ dxm1, const double* __restrict__ dxtm1){
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

      double rtmp,stmp,ttmp,wijke;
      __shared__ double shdxm1[LX1*LY1];
      __shared__ double shu[LX1*LY1];
      __shared__ double shur[LX1*LY1];
      __shared__ double shus[LX1*LY1];
      double ru[LZ1];
      double rw[LZ1];
      double rut;
      double G00,G01,G02,G11,G12,G22;
      int l,e,i,j,k,ijk,ij,ele;

      e = blockIdx.x;
      j = threadIdx.y;
      i = threadIdx.x;
      ij = i + j*LX1;
      ele = e*LX1*LY1*LZ1;

      shdxm1[ij] = dxm1[ij];
      #pragma unroll
      for( k = 0; k < LZ1; ++k){
        ru[k] = u[ij + k*LX1*LY1 + ele];
        rw[k] = 0.0;
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();
      #pragma unroll
      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1; 
        G00 = gxyz[ijk+0*LX1*LY1*LZ1+ele*6];
        G01 = gxyz[ijk+1*LX1*LY1*LZ1+ele*6];
        G02 = gxyz[ijk+2*LX1*LY1*LZ1+ele*6]; 
        G11 = gxyz[ijk+3*LX1*LY1*LZ1+ele*6];
        G12 = gxyz[ijk+4*LX1*LY1*LZ1+ele*6];
        G22 = gxyz[ijk+5*LX1*LY1*LZ1+ele*6];
        ttmp = 0.0;
        shu[ij] = ru[k];
        for (l = 0; l<LX1; l++){
          ttmp += shdxm1[k+l*LX1] * ru[l];
        }
        __syncthreads();
 
        rtmp = 0.0;
        stmp = 0.0;
        #pragma unroll
        for (l = 0; l<LX1; l++){
          rtmp += shdxm1[i+l*LX1] * shu[l+j*LX1];
          stmp += shdxm1[j+l*LX1] * shu[i+l*LX1];
        }
        shur[ij] = G00*rtmp
                 + G01*stmp
                 + G02*ttmp;
        rut      = G02*rtmp
                 + G12*stmp 
                 + G22*ttmp;
        shus[ij] = G01*rtmp
                 + G11*stmp
                 + G12*ttmp;

      __syncthreads();

        wijke = 0.0;
        #pragma unroll
        for (l = 0; l<LX1; l++){
          wijke += shdxm1[l + i*LX1] * shur[l+j*LX1];
          rw[l] += shdxm1[k+l*LX1] * rut; 
          wijke += shdxm1[l + j*LX1] * shus[i+l*LX1];
        }
        rw[k] += wijke;
      }
      #pragma unroll
      for (k=0; k<LZ1; ++k){
        w[ij + k*LX1*LY1 + ele] = rw[k]; 
      }
}
extern "C" {
  void ax_cuda2_(double* __restrict__ w, const double* __restrict__ u, const double* __restrict__ gxyz,
		 const double* __restrict__ dxm1, const double* __restrict__ dxtm1, const int *nel){
    ax_cuda2_kernel<<<*nel,dim3(LX1,LY1,1)>>>(w, u, gxyz, dxm1, dxtm1);
  }
}
